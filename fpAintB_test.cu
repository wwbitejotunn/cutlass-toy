#include "hip/hip_runtime.h"
#include "cutlass_fpAintB/cutlass_kernels/fpA_intB_gemm/fpA_intB_gemm_template.h"
#include "stdlib.h"
#include <chrono>
#include "iostream"
#include <string>
namespace fastertransformer{
template class CutlassFpAIntBGemmRunner<half, uint8_t>;
}  // namespace fastertransformer

int main(int argc, char *argv[]){
    // m n k
    // argv[1], argv[2], argv[3]
    int m = strtol(argv[1], nullptr, 0);
    int n = strtol(argv[2], nullptr, 0);
    int k = strtol(argv[3], nullptr, 0);

    const auto kWarmTime=3;
    const auto kTestTime=10;

    auto mixed_gemm_runner = fastertransformer::CutlassFpAIntBGemmRunner<half, uint8_t>();

    int mixgemm_max_size=std::max(m,k);
    int mixgemm_workspace_size_bytes=mixed_gemm_runner.getWorkspaceSize(m, mixgemm_max_size, mixgemm_max_size);
    char *mixgemm_workspace_data;
    hipMalloc(&mixgemm_workspace_data, mixgemm_workspace_size_bytes);
    std::vector<half> a_half(m*k);
    for(auto & i:a_half){
        i=(float)rand()/RAND_MAX*20.0;
    }
    std::vector<int8_t> b_int(k*n);
    for(auto & i:b_int){
        i=rand()%256-127;
    }
    std::vector<half> b_scale_half(n);
    for(auto & i:b_scale_half){
        i=(float)rand()/RAND_MAX*0.05;
    }
    std::vector<half> c_half(m*n);
    void* d_a_half;
    void* d_b_int;
    void* d_b_scale;
    void* d_c_half;
    hipMalloc(&d_a_half, m*k*sizeof(half));
    hipMalloc(&d_b_int, k*n*sizeof(int8_t));
    hipMalloc(&d_b_scale, n*sizeof(half));
    hipMalloc(&d_c_half, m*n*sizeof(half));
    hipMemcpy(d_a_half,a_half.data(),m*k*sizeof(half),hipMemcpyHostToDevice);
    hipMemcpy(d_b_int,b_int.data(),k*n*sizeof(int8_t),hipMemcpyHostToDevice);
    hipMemcpy(d_b_scale,b_scale_half.data(),n*sizeof(half),hipMemcpyHostToDevice);
    std::cout<<"=== do warm up for "<<kWarmTime<<" times"<<std::endl;
    for(int i=0;i<kWarmTime;i++){
        mixed_gemm_runner.gemm(
            reinterpret_cast<const half*>(d_a_half),
            reinterpret_cast<const uint8_t*>(d_b_int),
            reinterpret_cast<const half*>(d_b_scale),
            reinterpret_cast<half*>(d_c_half),
            m,
            n,
            k,
            mixgemm_workspace_data,
            mixgemm_workspace_size_bytes,
            0
        );
    }
    hipDeviceSynchronize();
    auto start = std::chrono::system_clock::now();    
    for(int i=0;i<kTestTime;i++){
        mixed_gemm_runner.gemm(
            reinterpret_cast<const half*>(d_a_half),
            reinterpret_cast<const uint8_t*>(d_b_int),
            reinterpret_cast<const half*>(d_b_scale),
            reinterpret_cast<half*>(d_c_half),
            m,
            n,
            k,
            mixgemm_workspace_data,
            mixgemm_workspace_size_bytes,
            0
        );
    }
    hipDeviceSynchronize();
    auto stop = std::chrono::system_clock::now();    
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>((stop - start));
    std::cout<<"avg time for "<<kTestTime<<" run:"<<duration.count()/kTestTime<<std::endl;
    return 0;
}